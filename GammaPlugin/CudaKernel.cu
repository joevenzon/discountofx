#include "hip/hip_runtime.h"
enum e_params
{
#define PARAMFLOAT(x, def, name, hint) _##x,
#define PARAMCOUNT(x) k_param_##x
#include "params.h"
#undef PARAMFLOAT
#undef PARAMCOUNT
};

__device__ __host__ float calcLuminance(float r, float g, float b)
{
    return r * 0.2126f + g * 0.7152f + b * 0.0722f;
}

__device__ __host__ float lerp(float x, float y, float s)
{
    return x + s*(y - x);
}

__device__ __host__ void InternalGainAdjustKernel(int x, int y, float* params, const float* p_Input, float* p_Output)
{
    /*float origLum = calcLuminance(p_Input[0], p_Input[1], p_Input[2]);

    float distanceR = p_Input[0] - origLum;
    float distanceG = p_Input[1] - origLum;
    float distanceB = p_Input[2] - origLum;

    p_Output[0] = origLum + pow(abs(distanceR), params[_exponent]) * params[_gain] * (distanceR >= 0 ? 1 : -1);
    p_Output[1] = origLum + pow(abs(distanceG), params[_exponent]) * params[_gain] * (distanceG >= 0 ? 1 : -1);
    p_Output[2] = origLum + pow(abs(distanceB), params[_exponent]) * params[_gain] * (distanceB >= 0 ? 1 : -1);*/

    p_Output[0] = pow(max(0.f, p_Input[0]), params[_gamma]);
    p_Output[1] = pow(max(0.f, p_Input[1]), params[_gamma]);
    p_Output[2] = pow(max(0.f, p_Input[2]), params[_gamma]);

    //float newLum = calcLuminance(p_Output[0], p_Output[1], p_Output[2]);
    //float makeup = lerp(1.0f, origLum / max(newLum,0.01f), params[_preserveLum]);

    /*p_Output[0] *= makeup;
    p_Output[1] *= makeup;
    p_Output[2] *= makeup;*/
    
    p_Output[3] = p_Input[3];
}

__global__ void GainAdjustKernel(int p_Width, int p_Height, float* p_Params, const float* p_Input, float* p_Output)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < p_Width) && (y < p_Height))
   {
       const int index = ((y * p_Width) + x) * 4;

       InternalGainAdjustKernel(x, y, p_Params, &p_Input[index], &p_Output[index]);
   }
}

void RunCudaKernel(void* p_Stream, int p_Width, int p_Height, float* p_Params, const float* p_Input, float* p_Output)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);
    hipStream_t stream = static_cast<hipStream_t>(p_Stream);

    const int paramBytes = sizeof(float) * k_param_count;
    float* d_Params;
    hipMalloc(&d_Params, paramBytes);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_Params, p_Params, paramBytes, hipMemcpyHostToDevice);

    GainAdjustKernel<<<blocks, threads, 0, stream>>>(p_Width, p_Height, d_Params, p_Input, p_Output);

    // Free device memory
    hipFree(d_Params);
}
